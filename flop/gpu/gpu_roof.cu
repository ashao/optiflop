#include "hip/hip_runtime.h"
#include "gpu_roof.h"

__global__ void saxpy(int n, float a, float *x, float *y)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) y[i] = a * x[i] + y[i];
}

extern "C"
void gpu_axpy(int n, float a, float b, float * x_in, float * y_in,
              struct roof_args *args)
{
    float *x, *y;
    dim3 threadBlockRows(256, 1);

    size_t nbytes;

    nbytes = n * sizeof(float);

    hipMalloc(&x, nbytes);
    hipMalloc(&y, nbytes);

    hipMemcpy(x, x_in, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(y, y_in, nbytes, hipMemcpyHostToDevice);

    saxpy<<<(n + 255)/256, 256>>>(n, a, x, y);

    hipMemcpy(y_in, y, nbytes, hipMemcpyDeviceToHost);

    hipFree(x);
    hipFree(y);
}
